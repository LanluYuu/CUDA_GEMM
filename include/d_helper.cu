#pragma once
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#define BLOCK_SIZE 4096
#define ELE_IDX(x, y, col) (x * col + y)
#define FLOAT4(arr) reinterpret_cast<float4*>(&arr)[0]

struct d_Matrix { 
    int32_t height;
    int32_t width;
    float* data;
    d_Matrix(int32_t row, int32_t col) : height(row), width(col) {
        hipError_t err = hipMalloc(&data, height * width * sizeof(float));
        if (err != hipSuccess || data == nullptr) {
            printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        }
        printf("new matrix addr:0x%lX", &data);
    }

    ~d_Matrix() {
        hipFree(data);
    }
};
//device function
__device__ float d_GetMatrixElement(const d_Matrix& A, int32_t row, int32_t col) {
    return A.data[ELE_IDX(row, col, A.width)];
}

__device__ void d_SetMatrixElement(d_Matrix& A, int32_t row, int32_t col, float val) {
    //printf("val:%f", val);
    A.data[ELE_IDX(row, col, A.width)] = val;
    printf("\nkernel C_d matrix data addr:0x%lX", &A.data);
    //printf(", C.data:%f\n", A.data[ELE_IDX(row, col, A.width)]);
    return;
}