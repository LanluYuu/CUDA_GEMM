#pragma once
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#define BLOCK_SIZE 32
#define ELE_IDX(x, y, col) (x * col + y)

struct d_Matrix { 
    int32_t height;
    int32_t width;
    float* data;

    d_Matrix(int32_t row, int32_t col) : height(row), width(col) {
        hipError_t err = hipMalloc(&data, height * width * sizeof(float));
        if (err != hipSuccess) {
            printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        }
    }

    ~d_Matrix() {
        hipFree(data);
    }
};
//device function
__device__ float d_GetMatrixElement(const d_Matrix& A, int32_t row, int32_t col) {
    return A.data[ELE_IDX(row, col, A.width)];
}

__device__ void d_SetMatrixElement(const d_Matrix& A, int32_t row, int32_t col, float val) {
    printf("val:%f", val);
    A.data[ELE_IDX(row, col, A.width)] = val;
    printf(", C.data:%f\n", A.data[ELE_IDX(row, col, A.width)]);
    return;
}