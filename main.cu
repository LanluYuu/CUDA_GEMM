#include "gemm_v0.cu"
#include "gemm_v1.cu"
#include "helper.h"

//host code
void MatMul(int32_t x, int32_t y, int32_t z) { //Matirx A(x, z) * B(z, y)
    //init Matrix
    Matrix A(x, z);
    Matrix B(z, y);
    Matrix C(x, y);
    Matrix C_ref(x, y);
    //generate random value for A matrix
    GenRdVal4Mat(A);
    GenRdVal4Mat(B);
    //compute golden by CPU
    ComputeGolden(A, B, C_ref);
    //init Matrix on device
    d_Matrix A_d(x, z);
    d_Matrix B_d(z, y);
    d_Matrix C_d(x, y);
    //memcpy to device
    size_t A_size = A.height * A.width * sizeof(float);
    size_t B_size = B.height * B.width * sizeof(float);
    size_t C_size = C.height * C.width * sizeof(float);
    hipMemset(C_d.data, 1, C_size);
    hipMemcpy(A_d.data, A.data, A_size, hipMemcpyHostToDevice);
    hipMemcpy(B_d.data, B.data, B_size, hipMemcpyHostToDevice);
    //invoke kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // =====v0=====
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((x - 1) / BLOCK_SIZE + 1, (y - 1) / BLOCK_SIZE + 1);
    // gemm_v0<<<dimGrid, dimBlock>>> (A_d, B_d, C_d);
    // =====v0===== 
    // =====v1=====
    //dim3 dimBlock(BLOCK_SIZE / 2, BLOCK_SIZE / 2);
    //dim3 dimGrid((x - 1) / BLOCK_SIZE + 1, (y - 1) / BLOCK_SIZE + 1);
    // =====v1=====
    hipEventRecord(start, 0);
    gemm_v0<<<dimGrid, dimBlock>>> (A_d, B_d, C_d);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //memcpy to host
    hipError_t err = hipMemcpy(A_d.data, C.data, C_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
    }
    printf("C[0][0]:%f", C.data[0]);
    //compare
    if (CompareMat(C, C_ref)) {
        printf("\nresult pass!\n");
    } else {
        printf("\ncompare fail!\n");
    }
    std::cout << "\nexecute time:" << milliseconds << "ms" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    MatMul(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
}