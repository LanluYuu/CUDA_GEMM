#include "gemm_v0.cu"
#include "gemm_v1.cu"
#include "gemm_v2.cu"
#include "gemm_v3.cu"
#include "gemm_v4.cu"
#include "ref_cublas.cu"
#include "helper.h"

//host code
void MatMul(int32_t m, int32_t k, int32_t n) { //Matirx A(m, k) * B(k, n)
    //init Matrix
    Matrix A(m, k);
    Matrix B(k, n);
    Matrix C(m, n);
    Matrix C_ref(m, n);
    printf("init host matirx done\n");
    //generate random value for A matrix
    GenRdVal4Mat(A);
    GenRdVal4Mat(B);
    hipError_t err = hipSetDevice(1);
    if (err != hipSuccess) {
        std::cerr << "\nInit device failed!\n" << hipGetErrorString(err) << std::endl;
    }
    //set device 
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "device name:" << prop.name << std::endl;

    //init Matrix on device
    float *A_d; float *B_d; float *C_d; float *C_d_ref;
    size_t A_size = m * k * sizeof(float);
    size_t B_size = k * n * sizeof(float);
    size_t C_size = m * n * sizeof(float);
    hipMalloc(&A_d, A_size);
    hipMalloc(&B_d, B_size);
    hipMalloc(&C_d, C_size);
    hipMalloc(&C_d_ref, C_size);
    //memcpy to device
    err = hipMemcpy(A_d, A.data, A_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
    }
    err = hipMemcpy(B_d, B.data, B_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
    }
    //compute golden using cublas
    computeGoldenBlas(A_d, B_d, C_d_ref, C_ref.data, m, k, n);
    //invoke kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // =====v0=====
#if K_VERSION == 0
    dim3 dimBlock(32, 32); //set threads per block
    dim3 dimGrid((m - 1) / dimBlock.x + 1, (n - 1) / dimBlock.y + 1);
#elif K_VERSION == 1
    // =====v1=====
    dim3 dimBlock(32, 32); //set threads per block
    dim3 dimGrid((m - 1) / dimBlock.x + 1, ((n - 1) / dimBlock.y + 1) / 8);
#elif K_VERSION == 2
    // =====v1.5=====
    dim3 dimBlock(64, 8); //set threads per block
    dim3 dimGrid((m - 1) / dimBlock.x + 1, ((n - 1) / dimBlock.y + 1) / 8);
#elif K_VERSION == 3
    // =====v2=====
    dim3 dimBlock(16, 16);
    dim3 dimGrid(((m - 1) / dimBlock.x + 1) / 8, ((n - 1) / dimBlock.y + 1) / 8);
    //dim3 dimGrid(1, 1);
#elif K_VERSION == 4
    // =====v2=====
    dim3 dimBlock(16, 16);
    dim3 dimGrid(((m - 1) / dimBlock.x + 1) / 8, ((n - 1) / dimBlock.y + 1) / 8);
#endif
    //warm up for 10times
    for (int32_t i = 0; i < WARMUPT; ++i) {
#if K_VERSION == 0
        gemm_v0<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#elif K_VERSION == 1
        gemm_v1<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#elif K_VERSION == 2
        gemm_v2_1<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#elif K_VERSION == 3
        gemm_v3_1<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#elif K_VERSION == 4
        gemm_v4<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#endif
        hipDeviceSynchronize();
    }
    // =====v1=====
    hipEventRecord(start, 0);
#if K_VERSION == 0
    gemm_v0<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#elif K_VERSION == 1
    gemm_v1<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#elif K_VERSION == 2
    gemm_v2_1<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#elif K_VERSION == 3
    gemm_v3_1<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#elif K_VERSION == 4
        gemm_v4<<<dimGrid, dimBlock>>> (A_d, B_d, C_d, m, k, n);
#endif
    printf(" compute gemm done\n");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //memcpy to host
    err = hipMemcpy(C.data, C_d, C_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
    }
    //compare
    //std::cout << "C data:" << C.data[0] << "," << C.data[1] << "," << C.data[2] << std::endl;
    //std::cout << "C ref data:" << C_ref.data[0] << "," << C_ref.data[1] << "," << C_ref.data[2] << std::endl;
    if (CompareMat(C, C_ref)) {
        printf("\nresult pass!\n");
    } else {
        printf("\ncompare fail!\n");
    }
    std::cout << "\nExecute time:" << milliseconds << "ms" << std::endl;
    double flopsPerMairixMul = 2.0 * k * m * n;
    double tflops = (flopsPerMairixMul * 1e-12) / (milliseconds * 1e-3);
    std::cout << "Throuphput:" << tflops << "TFLOPS\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    MatMul(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
}