#define CUBLASAPI
#include <hipblas.h>
#include "d_helper.cu"

__host__ bool computeGoldenBlas(float* A, float* B, float* C, float* C_ref, int32_t m, int32_t k, int32_t n) {
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("\ncublas handle create fail!\n");
        return false;
    }
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    float alpha = 1.0f;
    float beta  = 0.0f;

    //warmUp 10 times
    for (int32_t i = 0; i < 10; ++i) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                n, m, k,
                &alpha,
                B, n,
                A, k, 
                &beta, 
                C, n);
    }

    const int32_t run_times = 100;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    #pragma unroll
    for (int32_t times = 0; times < run_times; ++times) {    

        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    n, m, k,
                    &alpha,
                    B, n,
                    A, k, 
                    &beta, 
                    C, n);
    }
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float ms_sum = 0.0f;
    hipEventElapsedTime(&ms_sum, start, end);
    float avg_ms = ms_sum / run_times;
    printf("\nCublas Execute time:%fms\n", avg_ms);
    double flopsPerMairixMul = 2.0 * k * m * n;
    printf("Cublas Throuphput:%lfTFLOPS\n", (flopsPerMairixMul * 1.0e-12f) / (avg_ms * 1.0e-3f));
    size_t size = m * n * sizeof(float);
    hipMemcpy(C_ref, C, size, hipMemcpyDeviceToHost);
    
    hipEventDestroy(start);
    hipEventDestroy(end);
    
    hipblasDestroy(handle);
    
    return true;
}