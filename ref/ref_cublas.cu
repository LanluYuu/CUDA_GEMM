#define CUBLASAPI
#include <hipblas.h>
#include "d_helper.cu"

__host__ bool computeGoldenBlasFp16(half* A, half* B, half* C, half* C_ref, int32_t m, int32_t k, int32_t n) {
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("\ncublas handle create fail!\n");
        return false;
    }

    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH); // Tensor Core

    half alpha = half(1.0f);
    half beta  = half(0.0f);

    //warmUp 10 times
    for (int32_t i = 0; i < 10; ++i) {
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                n, m, k,
                &alpha,
                B, HIP_R_16F, n,
                A, HIP_R_16F, k, 
                &beta, 
                C, HIP_R_16F, n,
                HIP_R_16F, HIPBLAS_GEMM_DEFAULT);
    }

    const int32_t run_times = 100;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    #pragma unroll
    for (int32_t times = 0; times < run_times; ++times) {    
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
            n, m, k,
            &alpha,
            B, HIP_R_16F, n,
            A, HIP_R_16F, k, 
            &beta, 
            C, HIP_R_16F, n,
            HIP_R_16F, HIPBLAS_GEMM_DEFAULT);
    }
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float ms_sum = 0.0f;
    hipEventElapsedTime(&ms_sum, start, end);
    float avg_ms = ms_sum / run_times;
    printf("\nCublas Execute time:%fms\n", avg_ms);
    double flopsPerMairixMul = 2.0 * k * m * n;
    printf("Cublas Throuphput:%lfTFLOPS\n", (flopsPerMairixMul * 1.0e-12f) / (avg_ms * 1.0e-3f));
    size_t size = m * n * sizeof(half);
    hipMemcpy(C_ref, C, size, hipMemcpyDeviceToHost);
    
    hipEventDestroy(start);
    hipEventDestroy(end);
    
    hipblasDestroy(handle);
    
    return true;
}

__host__ bool computeGoldenBlasFp32(float* A, float* B, float* C, float* C_ref, int32_t m, int32_t k, int32_t n) {
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("\ncublas handle create fail!\n");
        return false;
    }

    float alpha = 1.0f;
    float beta  = 0.0f;

    //warmUp 10 times
    for (int32_t i = 0; i < 10; ++i) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                n, m, k,
                &alpha,
                B, n,
                A, k, 
                &beta, 
                C, n);
    }

    const int32_t run_times = 100;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    #pragma unroll
    for (int32_t times = 0; times < run_times; ++times) {    

        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    n, m, k,
                    &alpha,
                    B, n,
                    A, k, 
                    &beta, 
                    C, n);
    }
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float ms_sum = 0.0f;
    hipEventElapsedTime(&ms_sum, start, end);
    float avg_ms = ms_sum / run_times;
    printf("\nCublas Execute time:%fms\n", avg_ms);
    double flopsPerMairixMul = 2.0 * k * m * n;
    printf("Cublas Throuphput:%lfTFLOPS\n", (flopsPerMairixMul * 1.0e-12f) / (avg_ms * 1.0e-3f));
    size_t size = m * n * sizeof(float);
    hipMemcpy(C_ref, C, size, hipMemcpyDeviceToHost);
    
    hipEventDestroy(start);
    hipEventDestroy(end);
    
    hipblasDestroy(handle);
    
    return true;
}