#define CUBLASAPI
#include <hipblas.h>
#include "d_helper.cu"

__host__ bool computeGoldenBlas(float* A, float* B, float* C, float* C_ref, int32_t m, int32_t k, int32_t n) {
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("\ncublas handle create fail!\n");
        return false;
    }
    float alpha = 1.0f;
    float beta  = 0.0f;

    //warmUp 10 times
    for (int32_t i = 0; i < 10; ++i) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                n, m, k,
                &alpha,
                B, n,
                A, k, 
                &beta, 
                C, n);
    }
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                n, m, k,
                &alpha,
                B, n,
                A, k, 
                &beta, 
                C, n);
    
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, end);
    printf("\nExecute time:%fms\n", milliseconds);
    double flopsPerMairixMul = 2.0 * k * m * n;
    printf("Throuphput:%lfTFLOPS\n", (flopsPerMairixMul * 1.0e-12f) / (milliseconds * 1.0e-3f));
    size_t size = m * n * sizeof(float);
    hipMemcpy(C_ref, C, size, hipMemcpyDeviceToHost);
    
    hipEventDestroy(start);
    hipEventDestroy(end);
    
    hipblasDestroy(handle);
    
    return true;
}