#include <hip/hip_runtime.h>
#include <iostream>
__global__ void gemm_v0(float* A, float* B) {
    int32_t idx = threadIdx.x;
    
    B[idx] = A[idx];
}

void MatMul() { //Matirx A(x, z) * B(z, y)
    float A[] = {1, 1, 1, 1};
    float B[] = {2, 2, 2, 2};
    hipSetDevice(0);
    float* A_d; float* B_d;
    hipMalloc(&A_d, 4 * sizeof(float));
    hipMalloc(&B_d, 4 * sizeof(float));
    hipError_t err = hipMemcpy(A_d, A, 4 * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "\nInit device failed!\n" << hipGetErrorString(err) << std::endl;
    }
    dim3 dimGrid(1);
    dim3 dimBlock(4);
    gemm_v0<<<dimGrid, dimBlock>>> (A_d, B_d);
    err = hipMemcpy(B, B_d, 4 * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "\nInit device failed!\n" << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();
    hipFree(A_d);
    hipFree(B_d);

    for (int32_t i = 0; i < 4; ++i) {
        printf("%f,", B[i]);
    }
}

int main() {
    MatMul();
}